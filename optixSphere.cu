#include "hip/hip_runtime.h"
#include <optix.h>

#include "optixSphere.h"
#include <cuda/helpers.h>
#include <optix_device.h>

#include <sutil/vec_math.h>
#include <random.h>

#include <stdio.h>

#define RANDFLOAT3 make_float3(rnd(seed), rnd(seed), rnd(seed))

// Declare a constant Params structure that will be filled in by the host (CPU) before launch,
// and can be accessed by all the device (GPU) kernels.
extern "C" {
    __constant__ Params params;
}


// This utility function performs the actual ray tracing from the given origin in the given direction.
// If the ray intersects an object, the payload (prd) contains the shading information which is set in closest-hit or miss program.
static __forceinline__ __device__ void trace(
    OptixTraversableHandle handle,          // The traversable handle representing the scene to trace against
    float3                 ray_origin,      // The origin of the ray
    float3                 ray_direction,   // The direction of the ray
    float                  tmin,            // The minimum distance along the ray to check for intersections
    float                  tmax,            // The maximum distance along the ray to check for intersections
    Payload* prd                             // The payload registered with the ray
)
{
    // Convert the payload data to 32-bit float values that can be used in the optixTrace call
    unsigned int p0, p1, p2, o1, o2, o3, d1, d2, d3, a1, a2, a3, h, seed;

    p0 = __float_as_uint(prd->result.x);
    p1 = __float_as_uint(prd->result.y);
    p2 = __float_as_uint(prd->result.z);

    o1 = __float_as_uint(prd->origin.x);
    o2 = __float_as_uint(prd->origin.y);
    o3 = __float_as_uint(prd->origin.z);

    d1 = __float_as_uint(prd->direction.x);
    d2 = __float_as_uint(prd->direction.y);
    d3 = __float_as_uint(prd->direction.z);

    a1 = __float_as_uint(prd->attenuation.x);
    a2 = __float_as_uint(prd->attenuation.y);
    a3 = __float_as_uint(prd->attenuation.z);

    h = prd->hit;

    seed = prd->seed;

    // Perform the trace call which will call into the intersect, any-hit and closest-hit programs
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                   // rayTime: a value to simulate motion blur (not used here)
        OptixVisibilityMask(1), // Visibility mask to define which objects this ray should intersect
        OPTIX_RAY_FLAG_NONE,    // A set of flags that can be used to control ray behavior
        0,                      // SBT offset: Index into the Shader Binding Table
        1,                      // SBT stride: The step between records in the SBT used for consecutive rays
        0,                      // missSBTIndex: Index of the miss shader in the SBT
        p0, p1, p2,
        o1, o2, o3,
        d1, d2, d3,
        a1, a2, a3,
        h, seed           // payload
    );

    // Convert the 32-bit float values back to the payload data
    prd->result.x = __uint_as_float(p0);
    prd->result.y = __uint_as_float(p1);
    prd->result.z = __uint_as_float(p2);

    prd->origin.x = __uint_as_float(o1);
    prd->origin.y = __uint_as_float(o2);
    prd->origin.z = __uint_as_float(o3);

    prd->direction.x = __uint_as_float(d1);
    prd->direction.y = __uint_as_float(d2);
    prd->direction.z = __uint_as_float(d3);

    prd->attenuation.x = __uint_as_float(a1);
    prd->attenuation.y = __uint_as_float(a2);
    prd->attenuation.z = __uint_as_float(a3);

    prd->hit = h;

    prd->seed = seed;
}


// A helper function to set the payload for the current ray
static __forceinline__ __device__ void setPayload(Payload p)
{
    optixSetPayload_0(__float_as_uint(p.result.x));
    optixSetPayload_1(__float_as_uint(p.result.y));
    optixSetPayload_2(__float_as_uint(p.result.z));

    optixSetPayload_3(__float_as_uint(p.origin.x));
    optixSetPayload_4(__float_as_uint(p.origin.y));
    optixSetPayload_5(__float_as_uint(p.origin.z));

    optixSetPayload_6(__float_as_uint(p.direction.x));
    optixSetPayload_7(__float_as_uint(p.direction.y));
    optixSetPayload_8(__float_as_uint(p.direction.z));

    optixSetPayload_9(__float_as_uint(p.attenuation.x));
    optixSetPayload_10(__float_as_uint(p.attenuation.y));
    optixSetPayload_11(__float_as_uint(p.attenuation.z));

    optixSetPayload_12(p.hit);

    optixSetPayload_13(p.seed); 
}

// A helper function to retrieve the payload for the current ray
static __forceinline__ __device__ Payload getPayload()
{
    return Payload{
        make_float3(
            __uint_as_float(optixGetPayload_0()),
            __uint_as_float(optixGetPayload_1()),
            __uint_as_float(optixGetPayload_2())),
make_float3(
            __uint_as_float(optixGetPayload_3()),
            __uint_as_float(optixGetPayload_4()),
            __uint_as_float(optixGetPayload_5())),
        make_float3(
            __uint_as_float(optixGetPayload_6()),
            __uint_as_float(optixGetPayload_7()),
            __uint_as_float(optixGetPayload_8())),

        make_float3(
            __uint_as_float(optixGetPayload_9()),
            __uint_as_float(optixGetPayload_10()),
            __uint_as_float(optixGetPayload_11())),

        optixGetPayload_12(),
        (int)optixGetPayload_13()
    };
}

// A helper function to generate a random 3D vector that is inside the unit sphere (i.e. length < 1.0f)
static __forceinline__ __device__ float3 random_in_unit_sphere(unsigned int seed) {
    float3 p;
    do {
        p = 2.0f * RANDFLOAT3 - make_float3(1, 1, 1);
    } while (p.x * p.x + p.y * p.y + p.z * p.z >= 1.0f);
    return p;
}

// Filmic tonemap operator
static __forceinline__ __device__ float3 tonemap(float3 x)
{
    // Coefficients of a rational polynomial fit to the ACES filmic tone mapping curve.
    float A = 0.15f;
    float B = 0.50f;
    float C = 0.10f;
    float D = 0.20f;
    float E = 0.02f;
    float F = 0.30f;

    return ((x * (A * x + C * B) + D * E) / (x * (A * x + B) + D * F)) - E / F;
}

// The ray generation program. This is called once per pixel, and its job is to generate the primary rays.
extern "C" __global__ void __raygen__rg()
{
    // Compute the launch index, which corresponds to the pixel indices in the image
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    // Get a pointer to the ray generation data stored in the Shader Binding Table
    const RayGenData* rtData = (RayGenData*)optixGetSbtDataPointer();

    // Get the camera basis vectors and eye position from the ray generation data.
    const float3      U = rtData->camera_u;
    const float3      V = rtData->camera_v;
    const float3      W = rtData->camera_w;

    // Initialize the payload data for this ray. This will be modified by the closest-hit or miss program
    Payload payload = Payload{
        make_float3(1.0f, 1.0f, 1.0f),
        make_float3(0.0f, 0.0f, 0.0f),
        make_float3(1.0f, 0.0f, 0.0f),
        make_float3(1.0f, 1.0f, 1.0f), 1, 1
    };

    // Set an initial color payload for the ray which might be modified by the closest-hit or miss program
    float3       payload_rgb = make_float3(0.5f, 0.5f, 0.5f);
    unsigned int seed = tea<4>(idx.y, idx.x);

    // Sample the pixel multiple times and average the results
    int sample_batch_count = 2000;
    for (size_t i = 0; i < sample_batch_count; i++)
    {
        // Generate a unique seed for the current sample batch
        seed = tea<4>(idx.y * 1600 + idx.x, i);
        payload.seed = seed;

        // Generate a random subpixel offset for anti-aliasing
        float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

        // Normalized device coordinates (NDC) are in the range [-1, 1] for both x and y
        float2 d = 2.0f * make_float2((idx.x + subpixel_jitter.x) / (dim.x), (idx.y + subpixel_jitter.y) / (dim.y)) - 1.0f;

        // Calculate the ray origin and direction for the current pixel
        float3 origin = rtData->cam_eye;
        float3 direction = normalize(d.x * U + d.y * V + W);
        
        // Reset the payload data for this ray (every iteration of the loop)
        payload.attenuation = make_float3(1.0f, 1.0f, 1.0f);
        payload.direction = direction;

        // Trace the ray into the scene
        int max_depth = 20;
        for (int q = 0; q < max_depth; q++)
        {
            trace(
                params.handle,
                origin,
                direction,
                0.0005f,  // tmin
                1e16f,  // tmax
                &payload);

            // If the ray did not hit anything, terminate the loop after accumulating the background color
            if (payload.hit == 0) {
                payload_rgb += make_float3(payload.result.x, payload.result.y, payload.result.z);
                break;
            }

            // If the ray hit something, accumulate the material color and generate a new ray in the reflected direction
            origin = payload.origin;
            direction = payload.direction;
        }
    }

    float exposure = 1.0f;

    // Apply exposure before tonemapping
    payload_rgb = payload_rgb / sample_batch_count * exp2(exposure); // Incorporate exposure

    // Apply filmic tonemapping to the HDR values
    payload_rgb = tonemap(payload_rgb);

    // Scale to [0, 1] range
    payload_rgb = clamp(payload_rgb, 0.0f, 1.0f);

    // Optional: Apply simple gamma correction post tonemapping
    float gamma = 2.2f;
    payload_rgb = make_float3(
        powf(payload_rgb.x, 1.0f / gamma),
        powf(payload_rgb.y, 1.0f / gamma),
        powf(payload_rgb.z, 1.0f / gamma));

    // Write the tonemapped and gamma-corrected pixel color to the image buffer
    params.image[idx.y * params.image_width + idx.x] = make_color(payload_rgb);
}

// Modified sigmoid function to create the skybox's gradient
static __forceinline__ __device__ float modified_sigmoid(float angle) {
    float exponent = (0.99f - angle) * 1100.f;
    float sigmoid = (19.8f / (1.f + expf(exponent))) + 0.2f;
    
    return sigmoid;
}

// Fresnel-Schlick implementation for specular reflection
static __forceinline__ __device__ float3 fresnelSchlick(float cosTheta, const float3& F0)
{
    return F0 + (make_float3(1.0f) - F0) * powf(1.0f - cosTheta, 5.0f);
}

// The miss program. This is called for any ray that does not hit geometry.
extern "C" __global__ void __miss__ms()
{
    // Get a pointer to the miss program data stored in the Shader Binding Table.
    MissData* rt_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    // Read the existing payload. This payload was set by the raygen program.
    Payload payload = getPayload();

    // Calculate the skybox color based on the ray direction
    float3 unit_direction = normalize(payload.direction);
    float angle_mult = dot(unit_direction, normalize(make_float3(0.9f, 1.0f, 3.0f))); 
    float3 c = modified_sigmoid(angle_mult) * make_float3(0.29f, 0.58f, 0.94f); 

    // Set the payload to the resulting skybox color after applying the attenuation
    float3 sampled_color = payload.attenuation * c; 

    setPayload(Payload{
        sampled_color,
        payload.origin, payload.direction, make_float3(0.0f), 0, payload.seed
    });
}

// The closest hit program. This is called when a ray hits the closest geometry.
extern "C" __global__ void __closesthit__ch()
{
    // Optix gives us information about the intersected object and hit distance
    float  t_hit = optixGetRayTmax();

    // The ray's origin and direction
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();

    // Other information such as primitive index, traversable handle and SBT GAS index
    const unsigned int           prim_idx = optixGetPrimitiveIndex();
    const OptixTraversableHandle gas = optixGetGASTraversableHandle();
    const unsigned int           sbtGASIndex = optixGetSbtGASIndex();

    // Define and retrieve the sphere's data (center and radius)
    float4 q;
    optixGetSphereData(gas, prim_idx, sbtGASIndex, 0.f, &q);
    
    // Read the existing payload. This payload was set by the raygen program.
    Payload p = getPayload();

    // Retrieve the current HitGroupData from the SBT
    HitGroupData* hit_group_data = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());

    // Compute the intersection point in world space
    float3 world_raypos = ray_orig + t_hit * ray_dir;
    // Transform the intersection point from world space to object space
    float3 obj_raypos = optixTransformPointFromWorldToObjectSpace(world_raypos);
    // Determine the object space normal, and then transform it back to world space
    float3 obj_normal = (obj_raypos - make_float3(q)) / q.w;
    // Normalize the normal vector after transforming it to world space
    float3 world_normal = normalize(optixTransformNormalFromObjectToWorldSpace(obj_normal));

    // Use the provided color and other properties from the hit group data
    const float3 diffuse_albedo = hit_group_data->color;
    const float3 specular_albedo = hit_group_data->specular;
    const float roughness = hit_group_data->roughness;

    // Calculate reflection direction
    float3 reflection_direction = reflect(-ray_dir, world_normal);
    reflection_direction = normalize(reflection_direction + roughness * normalize(random_in_unit_sphere(p.seed)));

    // Calculate the specular component
    float3 half_vector = normalize(- ray_dir + normalize(reflect(-ray_dir, world_normal))); // half_vector = "halfway vector"
    float NoH = dot(world_normal, half_vector); // NoH = "normal dot half_vector"

    // Compute the Fresnel-Schlick term
    float3 F0 = make_float3(0.04f); // Assume non-metallic
    float3 F = fresnelSchlick(fmaxf(dot(world_normal, -ray_dir), 0.0f), F0);

    // Combine the specular and diffuse components
    float3 specular_component = F * specular_albedo;
    float3 diffuse_component = (1.0f - F) * diffuse_albedo;

    // Combine the specular and diffuse components by adding them together
    float3 material_response = diffuse_component + specular_component;
    
    float3 target;
    float3 new_dir;

    unsigned int seed = p.seed;
    float random = rnd(seed);
    if (random > sqrt(dot(F, F))) {
        // Diffuse component
        target = world_raypos + world_normal + random_in_unit_sphere(p.seed);
        new_dir = normalize(target - world_raypos);
	} else {
		// Specular component
        target = world_raypos + reflection_direction;
		new_dir = normalize(target - world_raypos);
    }

    //float3 cur_attenuation = p.attenuation * (rt_data->color);
    float3 cur_attenuation = material_response * p.attenuation;

    // Set the payload to the resulting normal shaded color
    setPayload(Payload{
        make_float3(0.0f, 0.0f, 0.0f),
        make_float3(world_raypos.x, world_raypos.y, world_raypos.z),
        make_float3(new_dir.x, new_dir.y, new_dir.z), cur_attenuation, 1, p.seed
    });
}